#include "hip/hip_runtime.h"


template <typename T = float>
__global__ void read_feature_kernel(array_d_t<T> *dr, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

        uint64_t row_index = index_ptr[idx_idx] + key_off;
      	uint64_t tid = threadIdx.x % 32;


    for (; tid < dim; tid += 32) {
	    T temp = ptr[(row_index) * cache_dim + tid];
	    out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
    }
  }
}


template <typename T = float>
__global__ void SA_read_feature_kernel(SA_cache_d_t<T> *cache, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
    uint64_t row_index = index_ptr[idx_idx] + key_off;
    uint64_t tid = threadIdx.x % 32;

    cache->get_data(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim);

    // for (; tid < dim; tid += 32) {
    //   cache->get_data(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim);
    //   //   dr -> 
	  //   // T temp = ptr[(row_index) * cache_dim + tid];
	  //   // out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
    // }
  }
}


template <typename T = float>
__global__ void read_feature_kernel_with_cpu_backing_memory(array_d_t<T> *dr, range_d_t<T> *range, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, GIDS_CPU_buffer<T> CPU_buffer, bool cpu_seq, unsigned int* d_cpu_access, uint64_t key_off) {

  uint64_t bid = blockIdx.x;

  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

      uint64_t row_index = index_ptr[idx_idx] + key_off;
      uint64_t tid = threadIdx.x % 32;

      uint32_t cpu_off = range -> get_cpu_offset(row_index);


      if(cpu_seq){
        if(row_index < CPU_buffer.cpu_buffer_len){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);
          for (; tid < dim; tid += 32) {
            T temp = CPU_buffer.device_cpu_buffer[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
        for (; tid < dim; tid += 32) {
          T temp = ptr[(row_index) * cache_dim + tid];
          out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
        }
      }
      }
      else{
        if((cpu_off & 0x1) == 1){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);

            for (; tid < dim; tid += 32) {
              T temp = CPU_buffer.device_cpu_buffer[(cpu_off >> 1) * cache_dim + tid];
              out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
          for (; tid < dim; tid += 32) {
            T temp = ptr[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
          }
        }
      }
  }
}


template <typename T = float>
__global__ void set_cpu_buffer_kernel(range_d_t<T> *d_range, uint64_t* idx_ptr, int num, uint32_t pageSize) {
  
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx <  num){
    d_range -> set_cpu_buffer(idx_ptr[idx], idx );
  }
}


template <typename T = float>
__global__
void set_window_buffering_kernel(array_d_t<T>* dr, uint64_t *index_ptr, uint64_t page_size, int hash_off){
	bam_ptr<T> ptr(dr);
	if(threadIdx.x == 0){
		uint64_t page_idx = index_ptr[blockIdx.x] + hash_off;
		ptr.set_window_buffer_counter(page_idx * page_size/sizeof(T), 1);
	}
}

template <typename T = float>
__global__ void read_kernel(array_d_t<T> *dr,
                                    uint64_t num, uint64_t offset) {
      bam_ptr<T> ptr(dr);
     if(threadIdx.x == 0 && blockIdx.x == 0){
        for(uint64_t i = 0; i < num; i++){
              if(i == 0) printf("idx: %llu type size:%i \n", offset,  (int) sizeof(T));
             // T temp = ptr[i + offset];
              printf("read data: %llu\n",  (unsigned long long) ptr[i + offset]);
             // printf("float read data: %f\n", temp);

        }
     }                           
}

template <typename T = float>
__global__ void write_feature_kernel(Controller** ctrls, page_cache_d_t* pc, array_d_t<T> *dr, T* in_tensor_ptr,
                                    uint64_t num, uint64_t offset) {

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < num){
      bam_ptr<T> ptr(dr);
      ptr[idx + offset] = in_tensor_ptr[idx];
    }
}


template <typename T = float>
__global__ 
void
print_kernel(SA_cache_d_t<T> *cache){
  cache -> print_stats();
}
