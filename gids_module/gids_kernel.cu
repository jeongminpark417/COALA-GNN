#include "hip/hip_runtime.h"


template <typename T = float>
__global__ void read_feature_kernel(array_d_t<T> *dr, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;


  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

        uint64_t row_index = index_ptr[idx_idx] + key_off;
      	uint64_t tid = threadIdx.x % 32;


    for (; tid < dim; tid += 32) {
	    T temp = ptr[(row_index) * cache_dim + tid];
	    out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
    }
  }
}


template <typename T = float>
__global__ void SA_read_feature_kernel(SA_cache_d_t<T> *cache, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off, uint32_t head_ptr, uint8_t* static_info_ptr, uint8_t update_counter) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
    uint64_t row_index = index_ptr[idx_idx] + key_off;
    uint64_t tid = threadIdx.x % 32;

    cache->get_data(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim, head_ptr, idx_idx, static_info_ptr, update_counter);
  } 
}

template <typename T = float>
__global__ void SA_read_feature_kernel_with_PVP(SA_cache_d_t<T> *cache, T *out_tensor_ptr,
                                    int64_t *index_ptr, uint64_t** node_flag_ptr, T* PVP_pinned_data,  int dim,
                                    int64_t num_idx, int cache_dim, int cur_gpu, uint64_t key_off, uint32_t head_ptr, uint8_t* static_info_ptr, uint8_t update_counter, bool debug_mode, unsigned long long* debug_count = nullptr) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  uint64_t idx_idx = bid * num_warps + warp_id;
  //if(idx_idx == 0 && threadIdx.x == 0) printf("GPU ID:%llu num_idx size: %llu\n", (unsigned long long) (cache -> my_GPU_id_), (unsigned long long) num_idx);

  if (idx_idx < num_idx) {
    uint64_t row_index = index_ptr[idx_idx] + key_off;
    uint64_t tid = threadIdx.x % 32;


    uint64_t fetch_idx = node_flag_ptr[cur_gpu][idx_idx] ;

    //already prefetched
   if((fetch_idx >> 63) == (uint64_t) 1){
      //if(tid == 0) printf("prefetched KEY:%llu my GPU ID:%llu write GPU:%i IDX:%llu num_idx:%llu \n", row_index, (unsigned long long) (cache -> my_GPU_id_), cur_gpu,  (unsigned long long) idx_idx,(unsigned long long) num_idx);
      if(debug_mode) {
        if(tid == 0) atomicAdd(debug_count, 1);

      }
      // for(; tid < dim; tid += 32){
      //   uint64_t prefetched_idx = fetch_idx & (0x7FFFFFFFFFFFFFFF);
      //   out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = PVP_pinned_data[dim * fetch_idx + tid];
      // }
      T* PVP_data_ptr = PVP_pinned_data + dim * fetch_idx;
      cache->get_data_from_PVP(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim, head_ptr,idx_idx, static_info_ptr, update_counter, PVP_data_ptr);

    }

    else{
      cache->get_data(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim, head_ptr,idx_idx, static_info_ptr, update_counter);
    }
  }
  
}



template <typename T = float>
__global__ void read_feature_kernel_with_cpu_backing_memory(array_d_t<T> *dr, range_d_t<T> *range, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, GIDS_CPU_buffer<T> CPU_buffer, bool cpu_seq, unsigned int* d_cpu_access, uint64_t key_off) {

  uint64_t bid = blockIdx.x;

  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

      uint64_t row_index = index_ptr[idx_idx] + key_off;
      uint64_t tid = threadIdx.x % 32;

      uint32_t cpu_off = range -> get_cpu_offset(row_index);


      if(cpu_seq){
        if(row_index < CPU_buffer.cpu_buffer_len){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);
          for (; tid < dim; tid += 32) {
            T temp = CPU_buffer.device_cpu_buffer[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
        for (; tid < dim; tid += 32) {
          T temp = ptr[(row_index) * cache_dim + tid];
          out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
        }
      }
      }
      else{
        if((cpu_off & 0x1) == 1){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);

            for (; tid < dim; tid += 32) {
              T temp = CPU_buffer.device_cpu_buffer[(cpu_off >> 1) * cache_dim + tid];
              out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
          for (; tid < dim; tid += 32) {
            T temp = ptr[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
          }
        }
      }
  }
}


template <typename T = float>
__global__ void set_cpu_buffer_kernel(range_d_t<T> *d_range, uint64_t* idx_ptr, int num, uint32_t pageSize) {
  
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx <  num){
    d_range -> set_cpu_buffer(idx_ptr[idx], idx );
  }
}


template <typename T = float>
__global__
void set_window_buffering_kernel(array_d_t<T>* dr, uint64_t *index_ptr, uint64_t page_size, int hash_off){
	bam_ptr<T> ptr(dr);
	if(threadIdx.x == 0){
		uint64_t page_idx = index_ptr[blockIdx.x] + hash_off;
		ptr.set_window_buffer_counter(page_idx * page_size/sizeof(T), 1);
	}
}

template <typename T = float>
__global__ void read_kernel(array_d_t<T> *dr,
                                    uint64_t num, uint64_t offset) {
      bam_ptr<T> ptr(dr);
     if(threadIdx.x == 0 && blockIdx.x == 0){
        for(uint64_t i = 0; i < num; i++){
              if(i == 0) printf("idx: %llu type size:%i \n", offset,  (int) sizeof(T));
             // T temp = ptr[i + offset];
              printf("read data: %llu\n",  (unsigned long long) ptr[i + offset]);
             // printf("float read data: %f\n", temp);

        }
     }                           
}

template <typename T = float>
__global__ void write_feature_kernel(Controller** ctrls, page_cache_d_t* pc, array_d_t<T> *dr, T* in_tensor_ptr,
                                    uint64_t num, uint64_t offset) {

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < num){
      bam_ptr<T> ptr(dr);
      ptr[idx + offset] = in_tensor_ptr[idx];
    }
}


template <typename T = float>
__global__ void 
split_node_list_init_kernel(int64_t* index_ptr, uint64_t* index_pointer_list,  int64_t num_gpu,  int64_t index_size){
  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < index_size){
    int64_t cur_node = index_ptr[idx];
    int64_t gpu_id = cur_node % num_gpu;
    uint64_t counter_add = (index_pointer_list[gpu_id]);
    atomicAdd((unsigned int*) (index_pointer_list[gpu_id]), (unsigned int )1);
  }
}

template <typename T = float>
__global__ void 
split_node_list_kernel(int64_t* index_ptr, uint64_t* dist_index_ptr,  uint64_t* index_pointer_list,  int64_t num_gpu, int64_t index_size, 
    uint64_t* meta_buffer_ptr){
  
  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < index_size){
    int64_t cur_node = index_ptr[idx];
    int64_t gpu_id = cur_node % num_gpu;
    unsigned long long int enq_idx = atomicAdd((unsigned long long int*) (index_pointer_list[gpu_id]), (unsigned long long int)1);

    int64_t* dist_index = (int64_t*) (dist_index_ptr[gpu_id]);
    uint64_t* meta_buffer = (uint64_t*) (meta_buffer_ptr[gpu_id]);
    meta_buffer[enq_idx] = idx;
    dist_index[enq_idx] = cur_node;

  }

}


template <typename T = float>
 __forceinline__
__device__
void block_memcpy(void* dst, void* src, size_t size){
     T* src_ptr = (T*) src;
     T* dst_ptr = (T*) dst;
     
     uint32_t count = blockDim.x;     
     uint32_t my_id = threadIdx.x;

     for(; my_id < size; my_id += count){
          dst_ptr[my_id] =  src_ptr[my_id]; 
     }
 }



template <typename T = float>
__global__ 
void 
gather_feature_kernel(T *out_tensor_ptr, T* src_tensor_ptr, uint64_t* meta_buffer, int dim, int64_t num_idx, int rank, int my_rank){

  uint64_t r_idx = blockIdx.x;
  if(r_idx < num_idx){
    uint64_t dst_idx = meta_buffer[r_idx];
    // if(dst_idx == 1 && threadIdx.x == 0) {
    //   printf("my rank: %i src rank:%i r_idx:%llu src data 1: %f 2: %f\n", rank, my_rank, (unsigned long long) r_idx, (float) ((src_tensor_ptr + r_idx * dim)[0]),  (float) ((src_tensor_ptr + r_idx * dim)[1]));
    // }
    
    //block_memcpy<T>((void*) (out_tensor_ptr + dst_idx * dim), (void*)(src_tensor_ptr + r_idx * dim), dim * sizeof(T) / sizeof(T) );
    block_memcpy<uint64_t>((void*) (out_tensor_ptr + dst_idx * dim), (void*)(src_tensor_ptr + r_idx * dim), dim * sizeof(uint64_t) / sizeof(uint64_t) );

  }


}

template <typename T = float>
__global__ 
void
print_meta_buffer_kernel( uint64_t** d_meta_buffer, uint64_t gpu_id, uint64_t meta_len, uint64_t rank){
    uint64_t* meta_buffer = d_meta_buffer[gpu_id];
    for(int i = 0; i < meta_len; i++){
      printf("rank: %llu meta idx: %llu\n", (unsigned long long) rank, meta_buffer[i]);
    }

}




// Preemptive Victim-buffer Prefetcher
template <typename T = float>
__global__ 
void
update_reuse_counters_kernel(SA_cache_d_t<T> *cache, uint64_t** batch_arrays, uint64_t* batch_size_array, uint32_t num_gpus){
  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  uint64_t read_idx = bid * num_warps + warp_id;

  uint64_t y_bid = blockIdx.y;


  uint32_t reuse_time = (blockIdx.y / num_gpus);
  uint32_t GPU_id = blockIdx.y % num_gpus;
  const uint64_t num_idx = batch_size_array[y_bid];

 // if(bid == 0 && threadIdx.x ==0 && GPU_id != 0) printf("GPU id diff:%lu num_idx: %llu\n", (unsigned long)GPU_id, num_idx);
  //if(bid == 0 && threadIdx.x == 0) printf("reuse time:%i num_idx:%llu\n", (int) reuse_time, (unsigned long long) num_idx);

  if(read_idx < num_idx){
    uint64_t* index_ptr =(uint64_t*) (batch_arrays[y_bid]);
    uint64_t node_id = index_ptr[read_idx];
    //if(GPU_id != 0) printf("\t \t GPU_id correct\n");
    
    cache->update_reuse_val(node_id, reuse_time, GPU_id, read_idx,num_idx);
  }
  
}




template <typename T = float>
__global__ 
void fill_batch_kernel(uint64_t* PVP_pinned_idx, uint64_t** node_flag_ptr, uint32_t batch_size, int dim, bool debug_mode, unsigned  long long* debug_counter, unsigned int my_GPU, uint64_t max_sample_size) {


  uint64_t id = blockIdx.x * blockDim.x + threadIdx.x;
//  if(id == 0) printf("GPU: %llu batch size; %llu sizeof data:%llu\n",(unsigned long long)my_GPU, (unsigned long long) batch_size,  (unsigned long long)(sizeof(unsigned long long)));

  if (id < batch_size){
    uint64_t cur_idx =  PVP_pinned_idx[id];
    uint16_t cur_GPU_ID = (cur_idx >> 40) & 0x00FF;
    uint64_t batch_idx = (cur_idx & (0x000000FFFFFFFFFF));
    //printf("node ID Write id: %llu idx:%llu my GPU ID: %llu write GPU_id: %llu\n", id, batch_idx, (unsigned long long) my_GPU, (unsigned long long) cur_GPU_ID);
    uint64_t flag = id | 0x8000000000000000;

    if(batch_idx >=  max_sample_size || cur_GPU_ID > 1){
      printf("out of index GPU:%llu index: %llu GPU id: %llu id:%llu\n", (unsigned long long)my_GPU, (unsigned long long) batch_idx, (unsigned long long) cur_GPU_ID, (unsigned long long) id);
    }
      //printf("FILL GPU:%llu  index: %llu GPU id: %llu\n",(unsigned long long)my_GPU, (unsigned long long) batch_idx, (unsigned long long) cur_GPU_ID);

    node_flag_ptr[cur_GPU_ID][batch_idx] = (id | 0x8000000000000000);

    if(id == 0 && debug_mode) {
      atomicAdd(debug_counter, batch_size);
      //printf("batch_size: %llu\n", (unsigned long long) batch_size);
    }
    
  }

  else{

  }

}



__global__
void
get_static_info_kernel(uint8_t* out_ptr, uint64_t* index_ptr, uint64_t index_len, uint8_t* static_val_array ){
  uint64_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < index_len){
    uint64_t node_id  = index_ptr[id];
    uint8_t static_val = static_val_array[node_id];
    out_ptr[id] = static_val;
  }

}


template <typename T = float>
__global__ 
void
print_kernel(SA_cache_d_t<T> *cache, bool debug_mode, unsigned long long*  evict_counter, unsigned long long* prefetch_counter){
  cache -> print_stats();
  if(threadIdx.x == 0 && debug_mode) {
    printf("evict count: %llu\n", evict_counter[0]);
    printf("prefetch count: %llu\n", prefetch_counter[0]);

  }
}
