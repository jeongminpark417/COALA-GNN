#include "hip/hip_runtime.h"


template <typename T = float>
__global__ void read_feature_kernel(array_d_t<T> *dr, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;


  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

        uint64_t row_index = index_ptr[idx_idx] + key_off;
      	uint64_t tid = threadIdx.x % 32;


    for (; tid < dim; tid += 32) {
	    T temp = ptr[(row_index) * cache_dim + tid];
	    out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
    }
  }
}


template <typename T = float>
__global__ void SA_read_feature_kernel(SA_cache_d_t<T> *cache, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, uint64_t key_off) {

  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
    uint64_t row_index = index_ptr[idx_idx] + key_off;
    uint64_t tid = threadIdx.x % 32;

    cache->get_data(row_index, out_tensor_ptr + (bid * num_warps + warp_id) * dim);
  }
  
}


template <typename T = float>
__global__ void read_feature_kernel_with_cpu_backing_memory(array_d_t<T> *dr, range_d_t<T> *range, T *out_tensor_ptr,
                                    int64_t *index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, GIDS_CPU_buffer<T> CPU_buffer, bool cpu_seq, unsigned int* d_cpu_access, uint64_t key_off) {

  uint64_t bid = blockIdx.x;

  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;
  if (idx_idx < num_idx) {
 	    bam_ptr<T> ptr(dr);

      uint64_t row_index = index_ptr[idx_idx] + key_off;
      uint64_t tid = threadIdx.x % 32;

      uint32_t cpu_off = range -> get_cpu_offset(row_index);


      if(cpu_seq){
        if(row_index < CPU_buffer.cpu_buffer_len){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);
          for (; tid < dim; tid += 32) {
            T temp = CPU_buffer.device_cpu_buffer[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
        for (; tid < dim; tid += 32) {
          T temp = ptr[(row_index) * cache_dim + tid];
          out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
        }
      }
      }
      else{
        if((cpu_off & 0x1) == 1){
          if(tid == 0)
            atomicAdd(d_cpu_access, 1);

            for (; tid < dim; tid += 32) {
              T temp = CPU_buffer.device_cpu_buffer[(cpu_off >> 1) * cache_dim + tid];
              out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
            }
        }

        else{
          for (; tid < dim; tid += 32) {
            T temp = ptr[(row_index) * cache_dim + tid];
            out_tensor_ptr[(bid * num_warps + warp_id) * dim + tid] = temp;
          }
        }
      }
  }
}


template <typename T = float>
__global__ void set_cpu_buffer_kernel(range_d_t<T> *d_range, uint64_t* idx_ptr, int num, uint32_t pageSize) {
  
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx <  num){
    d_range -> set_cpu_buffer(idx_ptr[idx], idx );
  }
}


template <typename T = float>
__global__
void set_window_buffering_kernel(array_d_t<T>* dr, uint64_t *index_ptr, uint64_t page_size, int hash_off){
	bam_ptr<T> ptr(dr);
	if(threadIdx.x == 0){
		uint64_t page_idx = index_ptr[blockIdx.x] + hash_off;
		ptr.set_window_buffer_counter(page_idx * page_size/sizeof(T), 1);
	}
}

template <typename T = float>
__global__ void read_kernel(array_d_t<T> *dr,
                                    uint64_t num, uint64_t offset) {
      bam_ptr<T> ptr(dr);
     if(threadIdx.x == 0 && blockIdx.x == 0){
        for(uint64_t i = 0; i < num; i++){
              if(i == 0) printf("idx: %llu type size:%i \n", offset,  (int) sizeof(T));
             // T temp = ptr[i + offset];
              printf("read data: %llu\n",  (unsigned long long) ptr[i + offset]);
             // printf("float read data: %f\n", temp);

        }
     }                           
}

template <typename T = float>
__global__ void write_feature_kernel(Controller** ctrls, page_cache_d_t* pc, array_d_t<T> *dr, T* in_tensor_ptr,
                                    uint64_t num, uint64_t offset) {

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < num){
      bam_ptr<T> ptr(dr);
      ptr[idx + offset] = in_tensor_ptr[idx];
    }
}


template <typename T = float>
__global__ void 
split_node_list_init_kernel(int64_t* index_ptr, uint64_t* index_pointer_list,  int64_t num_gpu,  int64_t index_size){
  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < index_size){
    int64_t cur_node = index_ptr[idx];
    int64_t gpu_id = cur_node % num_gpu;
    uint64_t counter_add = (index_pointer_list[gpu_id]);
    atomicAdd((unsigned long long int*) (index_pointer_list[gpu_id]), (unsigned long long int)1);
  }
}

template <typename T = float>
__global__ void 
split_node_list_kernel(int64_t* index_ptr, uint64_t* dist_index_ptr,  uint64_t* index_pointer_list,  int64_t num_gpu, int64_t index_size, 
    uint64_t* meta_buffer_ptr){
  
  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < index_size){
    int64_t cur_node = index_ptr[idx];
    int64_t gpu_id = cur_node % num_gpu;
    unsigned long long int enq_idx = atomicAdd((unsigned long long int*) (index_pointer_list[gpu_id]), (unsigned long long int)1);

    int64_t* dist_index = (int64_t*) (dist_index_ptr[gpu_id]);
    uint64_t* meta_buffer = (uint64_t*) (meta_buffer_ptr[gpu_id]);
    meta_buffer[enq_idx] = idx;
    dist_index[enq_idx] = cur_node;

  }

}


template <typename T = float>
 __forceinline__
__device__
void block_memcpy(void* dst, void* src, size_t size){
     T* src_ptr = (T*) src;
     T* dst_ptr = (T*) dst;
     
     uint32_t count = blockDim.x;     
     uint32_t my_id = threadIdx.x;

     for(; my_id < size; my_id += count){
          dst_ptr[my_id] =  src_ptr[my_id]; 
     }
 }



template <typename T = float>
__global__ 
void 
gather_feature_kernel(T *out_tensor_ptr, T* src_tensor_ptr, uint64_t* meta_buffer, int dim, int64_t num_idx, int rank, int my_rank){

  uint64_t r_idx = blockIdx.x;
  if(r_idx < num_idx){
    uint64_t dst_idx = meta_buffer[r_idx];
    // if(dst_idx == 1 && threadIdx.x == 0) {
    //   printf("my rank: %i src rank:%i r_idx:%llu src data 1: %f 2: %f\n", rank, my_rank, (unsigned long long) r_idx, (float) ((src_tensor_ptr + r_idx * dim)[0]),  (float) ((src_tensor_ptr + r_idx * dim)[1]));
    // }
    block_memcpy<T>((void*) (out_tensor_ptr + dst_idx * dim), (void*)(src_tensor_ptr + r_idx * dim), dim * sizeof(T) / sizeof(T) );
  }


}

template <typename T = float>
__global__ 
void
print_meta_buffer_kernel( uint64_t** d_meta_buffer, uint64_t gpu_id, uint64_t meta_len, uint64_t rank){
    uint64_t* meta_buffer = d_meta_buffer[gpu_id];
    for(int i = 0; i < meta_len; i++){
      printf("rank: %llu meta idx: %llu\n", (unsigned long long) rank, meta_buffer[i]);
    }

}




// Preemptive Victim-buffer Prefetcher
template <typename T = float>
__global__ 
void
update_reuse_counters_kernel(SA_cache_d_t<T> *cache, uint64_t** batch_arrays, uint64_t* batch_size_array, uint32_t num_gpus){
  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int64_t read_idx = bid * num_warps + warp_id;

  uint64_t y_bid = blockIdx.y;


  uint32_t reuse_time = (blockIdx.y / num_gpus);
  uint32_t GPU_id = blockIdx.y % num_gpus;
  const uint64_t num_idx = batch_size_array[y_bid];

    
  //if(bid == 0 && threadIdx.x == 0) printf("reuse time:%i num_idx:%llu\n", (int) reuse_time, (unsigned long long) num_idx);

  if(read_idx < num_idx){
    uint64_t* index_ptr =(uint64_t*) (batch_arrays[y_bid]);
    uint64_t node_id = index_ptr[read_idx];
    
    cache->update_reuse_val(node_id, reuse_time, GPU_id, read_idx);
  }
  
}




template <typename T = float>
__global__ 
void
print_kernel(SA_cache_d_t<T> *cache){
  cache -> print_stats();
}
