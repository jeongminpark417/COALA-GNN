#include "hip/hip_runtime.h"
#include "nvshmem.h"
#include "nvshmemx.h"

__global__ 
void NVSHMEM_send_requests_kernel(int64_t *src_index_ptr, int64_t num_idx,int64_t * nvshmem_request_ptr, 
                                      int num_gpus, unsigned int* counters) {

  int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < num_idx){
    int64_t node_id = src_index_ptr[tid];
    int dest_pe_id = node_id % num_gpus;
    
    unsigned int dest_idx = atomicAdd(counters + dest_pe_id, 1) * 2;
    nvshmem_int64_p(nvshmem_request_ptr + dest_idx, node_id, dest_pe_id);
    nvshmem_int64_p(nvshmem_request_ptr + (dest_idx + 1), tid, dest_pe_id);
  }
}

__global__ 
void NVShmem_count_requests_kernel(int64_t* nvshmem_index_ptr, unsigned int* request_counters, uint64_t num_idx,  int n_gpus, int rank){
  
  int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < num_idx){
    int num_ph = blockDim.y / n_gpus;
    for(int ph = 0; ph < num_ph; ph++){
        int ydim = ph * n_gpus + threadIdx.y;
        int off = ydim * num_idx * 2;
        int64_t n_id = 0;
          n_id = nvshmem_int64_g(nvshmem_index_ptr + tid*2 + off, rank);
        if(n_id != -1){
          atomicAdd(request_counters + ydim, 1);
        }
    }
  }
}

template<typename Cache_Type>
__global__ 
void NVShmem_read_feature_kernel(int gpu_id, Cache_Type *cache, float *out_tensor_ptr,
                                    int64_t *nvshmem_index_ptr, int dim,
                                    int64_t num_idx, int cache_dim, int rank) {
  uint64_t bid = blockIdx.x;
  int num_warps = blockDim.x / 32;
  int warp_id = threadIdx.x / 32;
  int idx_idx = bid * num_warps + warp_id;

  if (idx_idx < num_idx) {
    //Request is a pair (node id, batch idx)
    int64_t row_index = nvshmem_index_ptr[idx_idx*2];
    int64_t batch_idx = nvshmem_index_ptr[idx_idx*2 + 1];
    uint64_t tid = threadIdx.x % 32;
    //    get_data(uint64_t id, T* output_ptr, int rank, int dst_gpu){

    cache->get_data(row_index, out_tensor_ptr + (batch_idx) * dim, rank, gpu_id);
  } 
}
